#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <stdio.h>
#include <c10/cuda/CUDAException.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define CUDA_ERR(ans) { gpuQssert((ans), __FILE__, __LINE__); }
inline void gpuQssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
__host__ __device__ inline unsigned int cdiv(unsigned int a, unsigned int b) { return (a+b-1)/b;}


// MatMul kernel
__global__ void fused_softmax_matmul_kernel(float* Q, float* K, float* O, int h, int w, int k) {
    const int TILE_SIZE = 32; // Define the tile size
    __shared__ float Qs[TILE_SIZE][TILE_SIZE];
    __shared__ float Ks[TILE_SIZE][TILE_SIZE];

    // Identify thread and map to matrix indices
    int rowIdx = blockIdx.y*blockDim.y + threadIdx.y;
    int colIdx = blockIdx.x*blockDim.x + threadIdx.x; 
    int rowIdxTile = threadIdx.y;
    int colIdxTile = threadIdx.x; 


    int nTilesHors = cdiv(w, TILE_SIZE);
    int nTileInns = cdiv(k, TILE_SIZE);

    float denominator = 0.0f;
    float numerator_exponent = 0.0f;
    float max_new = -INFINITY;
    float max_prev = -INFINITY;

    for (int nTilesHor=0; nTilesHor < nTilesHors; nTilesHor++) {
        float res_inn = 0.0f;

        for (int nTileInn=0; nTileInn < nTileInns; nTileInn++) {
            if (rowIdx < h && nTileInn * TILE_SIZE + colIdxTile < k) 
                Qs[rowIdxTile][colIdxTile] = Q[
                    rowIdx * k                         // Go to the right row
                    + nTileInn * TILE_SIZE + colIdxTile    // Iterate on the respecctive tile element in each tile
                ];
            else
                Qs[rowIdxTile][colIdxTile] = 0.0f;

            if (nTileInn * TILE_SIZE + rowIdxTile < k && nTilesHor*TILE_SIZE + colIdxTile < w)
                Ks[rowIdxTile][colIdxTile] = K[
                    w * (
                        nTileInn*TILE_SIZE   // number of rows to skip for the previuous tiles
                        + rowIdxTile    // number of rows to skip for the current tile
                    ) 
                    + nTilesHor*TILE_SIZE + colIdxTile                        // Go to the right column scanning horizontally
                ];
            else
                Ks[rowIdxTile][colIdxTile] = 0.0f;
            __syncthreads();
            
            for (int tile_k=0; tile_k< TILE_SIZE; tile_k++) {
                res_inn += Qs[rowIdxTile][tile_k] * Ks[tile_k][colIdxTile];
            };
            if (nTilesHor*TILE_SIZE + colIdxTile == colIdx) {
                numerator_exponent = res_inn;
            }
            __syncthreads();
        }
    
        max_new = fmaxf(max_prev, res_inn);
        if (max_new == max_prev) {
            denominator += __expf(res_inn - max_new);
        }
        else {
            denominator = denominator * __expf(max_prev - max_new) + 1.0f;
            max_new = max_prev;
        }

    }

    if (rowIdx < h && colIdx < w) {
        O[w*rowIdx + colIdx] = __expf(numerator_exponent - max_new)/denominator;
    };

}


torch::Tensor fused_softmax_matmul(torch::Tensor Q, torch::Tensor K) {
    const int TILE_SIZE = 32; // Define the tile size
    CHECK_INPUT(Q); CHECK_INPUT(K);
    const int h = Q.size(0);
    const int k = Q.size(1);
    const int w = K.size(1);

    assert (Q.size(1) == K.size(0) && "Matrix dimensions do not match for multiplication");
    torch::Tensor O = torch::zeros({h, w}, Q.options());


    dim3 tbp(TILE_SIZE, TILE_SIZE);
    dim3 blocks(cdiv(w, TILE_SIZE), cdiv(h, TILE_SIZE));
    fused_softmax_matmul_kernel<<<blocks, tbp>>>(
        Q.data_ptr<float>(), 
        K.data_ptr<float>(), 
        O.data_ptr<float>(), 
        h, 
        w,
        k
    );
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    return O;
}
